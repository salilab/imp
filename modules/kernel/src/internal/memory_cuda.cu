
#include <hip/hip_runtime.h>
/**
 *  \file IMP/kernel/src/internal/memory_cuda.cu
 *  \brief CUDA memory allocation functions
 *
 *  Copyright 2007-2024 IMP Inventors. All rights reserved.
 *
 */


#ifdef __NVCC__
#include <cuda_runtime.h>
#include <IMP/internal/memory_cuda.h>
#include <IMP/internal/cuda_helper_functions.h>

void *IMPcuda::kernel::internal::allocate_unified_cuda(std::size_t n) {
  void *p;
  IMP_checkCudaErrors(cudaMallocManaged(&p, n));
  return p;
}

void IMPcuda::kernel::internal::deallocate_unified_cuda(void *p) {
  if (p) {
    IMP_checkCudaErrors(cudaFree(p));
  }
}

#endif
