/**
 *  \file IMP/kernel/src/internal/random_number_generation_cuda.cu
 *  \brief GPU or CPU pooled random number generation
 *
 *  Copyright 2007-2022 IMP Inventors. All rights reserved.
 *
 */

#include <IMP/internal/random_number_generation_cuda.h>


#ifdef __NVCC__

/* Using updated (v2) interfaces to cublas and cusparse */
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <IMP/internal/cuda_helper_functions.h>

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>


static hiprandGenerator_t prngGPU; // GPU random number generator

void
IMPcuda::kernel::internal::get_random_numbers_normal_cuda
(double* p_random_array, unsigned int n,
 double mean, double stddev)
{
  double *d_Rand;
  IMP_checkCudaErrors(hipMalloc((void **)&d_Rand, n * sizeof(double)));
  IMP_checkCudaErrors(hiprandGenerateNormalDouble(prngGPU, d_Rand, n, mean, stddev));
  IMP_checkCudaErrors(hipMemcpy(p_random_array, d_Rand,
                                 n * sizeof(double), hipMemcpyDeviceToHost));
  IMP_checkCudaErrors(hipFree(d_Rand));
  IMP_checkCudaErrors(hipDeviceSynchronize()); // DEBUG
}

void
IMPcuda::kernel::internal::get_random_numbers_normal_cuda
(float* p_random_array, unsigned int n,
 float mean, float stddev)
{
  float *d_Rand;
  IMP_checkCudaErrors(hipMalloc((void **)&d_Rand, n * sizeof(float)));
  IMP_checkCudaErrors(hiprandGenerateNormal(prngGPU, d_Rand, n, mean, stddev));
  IMP_checkCudaErrors(hipMemcpy(p_random_array, d_Rand,
                                 n * sizeof(float), hipMemcpyDeviceToHost));
  IMP_checkCudaErrors(hipFree(d_Rand));
}

void
IMPcuda::kernel::internal::get_random_numbers_uniform_cuda
(float* p_random_array, unsigned int n)
{
  float *d_Rand;
  IMP_checkCudaErrors(hipMalloc((void **)&d_Rand, n * sizeof(float)));
  IMP_checkCudaErrors(hiprandGenerateUniform(prngGPU, d_Rand, n));
  IMP_checkCudaErrors(hipMemcpy(p_random_array, d_Rand,
                                 n * sizeof(float), hipMemcpyDeviceToHost));
  IMP_checkCudaErrors(hipFree(d_Rand));
}

void
IMPcuda::kernel::internal::get_random_numbers_uniform_cuda
(double* p_random_array, unsigned int n)
{
  double *d_Rand;
  IMP_checkCudaErrors(hipMalloc((void **)&d_Rand, n * sizeof(double)));
  IMP_checkCudaErrors(hiprandGenerateUniformDouble(prngGPU, d_Rand, n));
  IMP_checkCudaErrors(hipMemcpy
                      (p_random_array, d_Rand,
                       n * sizeof(double), hipMemcpyDeviceToHost));
  IMP_checkCudaErrors(hipFree(d_Rand));
}

bool IMPcuda::kernel::internal::init_gpu_rng_once
(unsigned long long seed, unsigned seed_counter, unsigned &last_seeded)
{
  static bool initialized(false); // is prngGPU initialized
  if(!initialized)
    {
      IMP_checkCudaErrors(hiprandCreateGenerator
                          (&prngGPU, HIPRAND_RNG_PSEUDO_MTGP32));
      IMP_checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(prngGPU, seed));
      last_seeded = seed_counter;
      initialized=true;
      return true; // success
    } else if (last_seeded != seed_counter) {
      // Simply setting the seed does not reset all internal state, so destroy
      // and recreate instead
      IMP_checkCudaErrors(hiprandDestroyGenerator(prngGPU));
      IMP_checkCudaErrors(hiprandCreateGenerator
                          (&prngGPU, HIPRAND_RNG_PSEUDO_MTGP32));
      IMP_checkCudaErrors(hiprandSetPseudoRandomGeneratorSeed(prngGPU, seed));
      last_seeded = seed_counter;
    }
  return false; // was already initialized
}

void
IMPcuda::kernel::internal::destroy_gpu_rng
()
{
  IMP_checkCudaErrors(hiprandDestroyGenerator(prngGPU));
}

#endif
