#include "hip/hip_runtime.h"
/**
 * \file cuda_helpers.cu
 * \brief GPU implementations of some SAXS operations
 *
 * Copyright 2007-2023 IMP Inventors. All rights reserved.
 *
 */

#include <IMP/internal/cuda_helper_functions.h>
#include <IMP/saxs/internal/cuda_helpers.h>
#include <iostream>
#include <cmath>
#include <stdio.h>

namespace IMPcuda {
namespace saxs {
namespace internal {

const static size_t MAX_THREADS = 512;

template <class T>
__device__ T square(T t) {
  return t * t;
}

__device__ double sinc_pi(double x) {
  if (x < 1e-6) {
    return 1.;
  } else {
    return sin(x) / x;
  }
}

__global__
void make_profile(const double *r_dist, const float *q, const double *distances,
                  float *intensity, double modulation_function_parameter,
                  size_t r_size, size_t q_size)
{
  __shared__ float inten[MAX_THREADS];
  // one block per q; each thread handles part of r_dist
  size_t k = blockIdx.x;
  inten[threadIdx.x] = 0.0;
  // iterate over radial distribution
  for (unsigned int r = threadIdx.x; r < r_size; r += blockDim.x) {
    double x = distances[r] * q[k];
    x = sinc_pi(x);
    // multiply by the value from distribution
    inten[threadIdx.x] += r_dist[r] * x;
  }
  __syncthreads();
  // get total intensity in first thread
  if (threadIdx.x == 0) {
    float total = 0.;
    for (size_t i = 0; i < blockDim.x; ++i) {
      total += inten[i];
    }
    // this correction is required since we approximate the form factor
    // as f(q) = f(0) * exp(-b*q^2)
    intensity[k] = total * std::exp(-modulation_function_parameter
                                    * square(q[k]));
  }
}

void squared_distribution_2_profile_cuda(
           const double *r_dist, const float *q,
           const double *distances, float *intensity,
           double modulation_function_parameter, size_t r_size, size_t q_size)
{
  double *d_r_dist, *d_distances;
  float *d_q, *d_intensity;

  IMP_checkCudaErrors(hipMalloc(&d_r_dist, r_size * sizeof(double)));
  IMP_checkCudaErrors(hipMalloc(&d_distances, r_size * sizeof(double)));
  IMP_checkCudaErrors(hipMalloc(&d_q, q_size * sizeof(float)));
  IMP_checkCudaErrors(hipMalloc(&d_intensity, q_size * sizeof(float)));

  // copy inputs to device
  IMP_checkCudaErrors(hipMemcpy(d_r_dist, r_dist, r_size * sizeof(double),
                                 hipMemcpyHostToDevice));
  IMP_checkCudaErrors(hipMemcpy(d_distances, distances,
                                 r_size * sizeof(double),
                                 hipMemcpyHostToDevice));
  IMP_checkCudaErrors(hipMemcpy(d_q, q, q_size * sizeof(float),
                                 hipMemcpyHostToDevice));

  size_t n_threads = std::min(MAX_THREADS, r_size);
  make_profile<<<q_size, n_threads>>>(d_r_dist, d_q, d_distances, d_intensity,
                                      modulation_function_parameter, r_size,
                                      q_size);
  IMP_checkCudaErrors(hipDeviceSynchronize());

  // copy outputs back to host
  IMP_checkCudaErrors(hipMemcpy(intensity, d_intensity, q_size * sizeof(float),
                                 hipMemcpyDeviceToHost));

  IMP_checkCudaErrors(hipFree(d_r_dist));
  IMP_checkCudaErrors(hipFree(d_distances));
  IMP_checkCudaErrors(hipFree(d_q));
  IMP_checkCudaErrors(hipFree(d_intensity));
}

} } }
